#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void cu_sobel(int *l_source_array_d, int *l_result_array_d, int rows,
                         int column_size) {
  int x_0, x_1, x_2, x_3, x_5, x_6, x_7, x_8, sum_0, sum_1, sum;
  int pos = blockIdx.x * column_size + threadIdx.x;
  int row = pos / column_size;
  int col = pos % column_size;

  bool top = (row == 0);
  bool bottom = (row == (rows - 1));
  bool left_edge = (col == 0);
  bool right_edge = (col == (column_size - 1));
  if (top == false && bottom == false && left_edge == false &&
      right_edge == false) {
    x_0 = l_source_array_d[(row - 1) * column_size + (col - 1)];
    x_1 = l_source_array_d[(row - 1) * column_size + (col)];
    x_2 = l_source_array_d[(row - 1) * column_size + (col + 1)];
    x_3 = l_source_array_d[(row)*column_size + (col - 1)];
    x_5 = l_source_array_d[(row)*column_size + (col + 1)];
    x_6 = l_source_array_d[(row + 1) * column_size + (col - 1)];
    x_7 = l_source_array_d[(row + 1) * column_size + (col)];
    x_8 = l_source_array_d[(row + 1) * column_size + (col + 1)];
    sum_0 = (x_0 + (2 * x_1) + x_2) - (x_6 + (2 * x_7) + x_8);
    sum_1 = (x_2 + (2 * x_5) + x_8) - (x_0 + (2 * x_3) + x_6);
    sum = sum_0 + sum_1;
    if (sum > 20) {
      sum = 255
    } else {
      sum = 0;
    }
    l_result_array_d[((row - 1) * (column_size - 2)) + (col - 1)] =
        sum_0 + sum_1;
  }
}

// Called from driver program.  Handles running GPU calculation
extern "C" void gpu_sobel(int *l_source_array, int *l_result_array,
                          int src_rows, int src_column_size) {
  int num_bytes_source = src_column_size * src_rows * sizeof(int);
  int *l_source_array_d;
  int *l_result_array_d;

  hipMalloc((void **)&l_source_array_d, num_bytes_source);
  hipMemcpy(l_source_array_d, l_source_array, num_bytes_source,
                       hipMemcpyHostToDevice);

  int result_column_size = src_column_size - 2;
  int result_row_size = src_rows - 2;
  int num_bytes_result = result_column_size * result_row_size * sizeof(int);
  hipMalloc((void **)&l_result_array_d, num_bytes_result);

  // block size should be adjusted to the problem size for performance
  dim3 block_size(src_column_size);
  // grid size should limit the amount of work to be completed
  dim3 grid_size(src_rows);

  cu_sobel<<<grid_size, block_size>>>(l_source_array_d, l_result_array_d,
                                      src_rows, src_column_size);

  // transfer results back to host
  hipMemcpy(l_result_array, l_result_array_d, num_bytes_result,
                       hipMemcpyDeviceToHost);

  // release the memory on the GPU
  hipFree(l_source_array_d);
  hipFree(l_result_array_d);
}
